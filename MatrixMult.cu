
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<time.h>
#define N 4

using namespace std;

__global__ void getMult(int *a,int *b,int *c,int n)
{
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	int col = blockIdx.y*blockDim.y + threadIdx.y;
	int sum=0;
	
	if(col<n && row<n)
	{
		for(int i=0;i<n;i++)
		{
			sum += a[row*n+i]*b[i*n+col];
		}
		c[row*n+col] = sum;
	}
}

int main()
{
	int a[N][N];
	int b[N][N];
	int c[N][N];
	
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			a[i][j] = rand()%N+1;
			b[i][j] = rand()%N+1;
			c[i][j] = 0;
		}
	}
	
	cout<<"Initial Status: "<<endl;
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			cout<<a[i][j]<<" ";
		}
		cout<<endl;
	}
	cout<<endl;
	
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			cout<<b[i][j]<<" ";
		}
		cout<<endl;
	}
	cout<<endl;
	
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			cout<<c[i][j]<<" ";
		}
		cout<<endl;
	}
	cout<<endl;
	
	
	int *d_a,*d_b,*d_c,*d_N;
	
	//Memory allocation
	hipMalloc(&d_a,N*N*sizeof(int));
	hipMalloc(&d_b,N*N*sizeof(int));
	hipMalloc(&d_c,N*N*sizeof(int));
	hipMalloc(&d_N,sizeof(int));
	
	//Memory Copy to device
	hipMemcpy(d_a,a,N*N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,N*N*sizeof(int),hipMemcpyHostToDevice);
	hipMemset(d_c,0,N*N*sizeof(int));
	//cudaMemcpy(d_N,N,sizeof(int),cudaMemcpyHostToDevice);
	
	dim3 dimGrid(1,1);
	dim3 dimBlock(16,16);
	
	getMult<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,N);
	hipDeviceSynchronize();
	
	hipMemcpy(c,d_c,N*N*sizeof(int),hipMemcpyDeviceToHost);
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			cout<<c[i][j]<<" ";
		}
		cout<<endl;
	}
	cout<<endl;
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_N);
	
	return 0;
}



