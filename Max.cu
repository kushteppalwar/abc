
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 2048

using namespace std;

__global__ void getMin(int input[])
{
	int tid = threadIdx.x;
	int step_count =1;
	int no_of_threads = blockDim.x;
	
	while(no_of_threads>0)
	{
		if(tid<no_of_threads)
		{
			int fst = tid*step_count*2;
			int snd = fst+step_count;
			
			if(input[fst]<input[snd])
				input[fst] = input[snd];
			else
				input[fst]=input[fst];
		}
		
		step_count <<=1;
		no_of_threads >>=1;
	}
	
}

int main()
{
	int *d_a;
	int *h_a;
	
	h_a = (int*)malloc(N*sizeof(int));
	hipMalloc(&d_a,N*sizeof(int));
	
	for(int i=0;i<N;i++)
	{
		//h_a[i] = rand()%N;
		h_a[i] = N-i+1;
	}
	
	//printing the array
	/*for(int i=0;i<N;i++)
	{
		printf("%d  ",h_a[i]);
	}*/
	
	hipMemcpy(d_a,h_a,N*sizeof(int),hipMemcpyHostToDevice);
	
	clock_t t1 = clock();
	getMin <<<1,N/2>>>(d_a);	
	hipDeviceSynchronize();
	clock_t t2 = clock()-t1;
	
	double time_taken;
	
	time_taken = ((double)t2)/CLOCKS_PER_SEC;
	
	int *result;
	result = (int*)malloc(sizeof(int));
	hipMemcpy(result,d_a,sizeof(int),hipMemcpyDeviceToHost);
	printf("Minimum number is: %d",result[0]);
	printf("Time taken is: %lf",time_taken);
	
	
	hipFree(d_a);
	free(result);
	free(h_a);
	
	return 0;
}
