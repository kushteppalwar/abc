
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 2048

using namespace std;


__global__ void FindSum(double input[])
{
	int tid = threadIdx.x;
	int step_size = 1;
	int no_of_threads = blockDim.x;
	
	while(no_of_threads>0)
	{
		if(tid<no_of_threads)
		{
			int fst = tid*step_size*2;
			 int snd = fst + step_size;
			input[fst]+=input[snd];
		}
		step_size <<=1;
		no_of_threads >>=1;
	}
}

int main()
{
	double *h_a;
	h_a = (double*)malloc(N*sizeof(double));
	double time_taken;
	
	
	//generating array
	for(int i=0;i<N;i++)
	{
		//h_a[i] = rand()%N;
		h_a[i] = i;
	}
	
	//printing Array
	for(int i=0;i<N;i++)
	{
		printf("%lf\n",h_a[i]);
	}
	printf("\n");
	
	double *d_a;
	hipMalloc(&d_a,N*sizeof(double));
	hipMemcpy(d_a,h_a,N*sizeof(double),hipMemcpyHostToDevice);
	
	clock_t t1 = clock();
	
	FindSum <<<1,N/2>>>(d_a);
	hipDeviceSynchronize();
	
	clock_t t2 = clock()-t1;
	
	double *result;
	result = (double*)malloc(sizeof(double));
	
	hipMemcpy(result,d_a,sizeof(double),hipMemcpyDeviceToHost);
	
	printf("\nSum: \t%lf",result[0]);
	time_taken = ((double)t2)/CLOCKS_PER_SEC;
	printf("Time Taken:%lf",time_taken);
	
	printf("\nSerial processing:\n");
	float res=0;
	
	clock_t t3 = clock();
	for(int i=0;i<N;i++)
	{
		res +=h_a[i];
	}
	clock_t t4 = clock()-t3;
	printf("Seq Result: %f",res);
	time_taken = ((double)t4)/CLOCKS_PER_SEC;
	printf("Time Taken:%lf",time_taken);
	
	hipFree(d_a);
	free(h_a);
	free(result);
	return 0;
	

}
