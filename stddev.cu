#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
#include<math.h>
#define N 16

using namespace std;


__global__ void FindSum(float input[])
{
	int tid = threadIdx.x;
	int step_count = 1;
	int no_of_threads = blockDim.x;
	
	while(no_of_threads>0)
	{
		
		if(tid < no_of_threads)
		{
			int fst = tid*step_count*2;
			int snd = fst + step_count;
			input[fst]+=input[snd];
		}
		
		step_count <<=1;
		no_of_threads >>=1;
	}
}

__global__ void FindDiff(float input[],float mean)
{
	int tid = threadIdx.x;

	while(tid<N)
	{
		input[tid] = input[tid] - mean;
	}		
}

int main()
{
	float *h_a;
	h_a = (float*)malloc(N*sizeof(float));
	
	for(int i=0;i<N;i++)
	{
		h_a[i] = ((float)rand()/RAND_MAX)*N;
	}
	
	for(int i=0;i<N;i++)
	{
		printf("%f\n",h_a[i]);
	}
	
	//Sum Calculations
	float *d_a;
	hipMalloc(&d_a,N*sizeof(float));
	hipMemcpy(d_a,h_a,N*sizeof(float),hipMemcpyHostToDevice);
	
	FindSum <<<1,N/2>>>(d_a);
	hipDeviceSynchronize();
	
	float *sum;
	sum = (float*)malloc(N*sizeof(float));
	hipMemcpy(sum,d_a,sizeof(float),hipMemcpyDeviceToHost);
	
	//mean
	float mean = (float)sum[0]/N;
	
	//Mean-Diff
	float *d_mean,*g;
	hipMalloc(&d_mean,sizeof(float));
	hipMalloc(&g,N*sizeof(float));
	
	hipMemcpy(d_mean,mean,sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(g,h_a,N*sizeof(float),hipMemcpyHostToDevice);
	
	
	FindDiff<<<1,N>>>(g,d_mean);
	hipDeviceSynchronize();
	
	float *dArray;
	dArray = (float*)malloc(N*sizeof(float));
	hipMemcpy(dArray,g,N*sizeof(float),hipMemcpyDeviceToHost);
	
	//Sum of Difference Array
	float *sDiff,*d_sArray;
	hipMalloc(&d_sArray,N*sizeof(float));
	sDiff = (float*)malloc(sizeof(float));
	
	FindSum <<<1,N/2>>>(d_sArray);
	hipDeviceSynchronize();
	
	hipMemcpy(sDiff,d_sArray,sizeof(float),hipMemcpyHostToDevice);
	
	float *temp;
	temp = (float*)malloc(sizeof(float));
	
	temp = (float)sDiff[0]/N;
	
	//stdDev
	
	float stdDev = sqrt(temp);
	
	printf("Standard Deviation: %f",stdDev);
	
	hipFree(d_a);
	hipFree(d_mean);
	hipFree(g);
	hipFree(d_sArray);
	free(h_a);
	free(temp);
	free(dArray);
	
	return 0;
}
	
