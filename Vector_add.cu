#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#define SIZE 50
using namespace std;

__global__ void add(int a[],int b[],int c[],int n)
{
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	int i = threadIdx.x;
	//if(i<n)
		c[i] = a[i]+b[i];
	
}

int main()
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	
	a = (int*)malloc(SIZE * sizeof(int));
	b = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));
	
	//Vector Generation
	for(int i=0;i<SIZE;i++)
	{
		a[i] = i + 1;
		b[i] = i;
	}

	hipMalloc(&d_a,SIZE * sizeof(int));
	hipMalloc(&d_b,SIZE * sizeof(int));
	hipMalloc(&d_c,SIZE * sizeof(int));
	
	hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	
	
	add<<<1,SIZE>>> (d_a,d_b,d_c,SIZE); 
	
	hipDeviceSynchronize(); //wait untill all the codes finish execution
	
	hipMemcpy(c,d_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	cout<<"\nVector A:\n";
	for(int i=0;i<SIZE;i++)
	{
		cout<<a[i]<<" ";
	}
	
	cout<<"\nVector B:\n";
	for(int i=0;i<SIZE;i++)
	{
		cout<<b[i]<<" ";
	}
	cout<<"\nVector SUM:\n";
	
	for(int i=0;i<SIZE;i++)
	{
		cout<<c[i]<<" ";
	}
	
	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}


